//CUDA version: 7.0
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <limits>
#include <chrono>

using namespace std;

hipError_t fWithCuda(float* f, float* x, int size);

__global__ void fKernel(float* dev_f, float* dev_x, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	dev_f[i] = sin(5 * powf(dev_x[i], 3)) + cos(7 + powf(dev_x[i], 4)) * dev_x[i];
}

int main()
{
	float a = 0.0f;
	float b = 100.0f;
	float h = 1e-4f;
    int arraySize = (b - a) / h;
	float* f = new float[arraySize];
	float* x = new float[arraySize];

	for (int i = 0; i < arraySize; i++){
		x[i] = a + i * h;
	}

	auto start = chrono::steady_clock::now();

	// calculate f function in parallel.
	hipError_t cudaStatus = fWithCuda(f, x, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	float min_x = a;
	float min_y = numeric_limits<float>::max();

	for (int i = 0; i < arraySize; i++){
		if (f[i] < min_y){
			min_y = f[i];
			min_x = x[i];
		}
	}

	auto end = chrono::steady_clock::now();
	float time_elapsed = float((end - start).count()) *
		chrono::steady_clock::period::num /
		chrono::steady_clock::period::den;

	fprintf(stdout, "CUDA result: %f %f\n", min_x, min_y);
	fprintf(stdout, "time elapsed: %f seconds\n", time_elapsed);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t fWithCuda(float* f, float* x, int size)
{
	float* dev_f = 0;
	float* dev_x = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for two vectors (one input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_f, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_x, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vector from host memory to GPU buffer.
	cudaStatus = hipMemcpy(dev_x, x, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element of array.
	fKernel <<<dim3(size / 500, 1), dim3(500, 1)>>> (dev_f, dev_x, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(f, dev_f, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
	hipFree(dev_f);
	hipFree(dev_x);
    
    return cudaStatus;
}
